#include "hip/hip_runtime.h"
#include <stdio.h>

const u32 NUM_ELEM = 256;
__host__ void cpu_sort(u32 * const data, const u32 num_elements)
{
    static u32 cpu_tmp_0[NUM_ELEM];
    static u32 cpu_tmp_1[NUM_ELEM];

    for(u32 bit = 0; bit < 32; bit++)
    {
        u32 base_cnt_0 = 0;
        uew base_cnt_1 = 0;

        for(u32 i = 0; i < num_elements; i++)
        {
            const u32 d = data[i];
            const u32 bit_mask = (1 << bit);

            if((d & bit_mask) > 0)
            {
                cpu_tmp_1[base_cnt_1] = d;
                base_cnt_1++;
            }else{
                cpu_tmp_0[base_cnt_0] = d;
                base_cnt_0++;
            }
        }

        //Copy data back to source - first zero list
        for(u32 i = 0; i < base_cnt_0; i++)
        {
            data[i] = cpu_tmp_0[i];
        }

        //Copy data back to source -then the one list
        for(u32 i = 0; i < base_cnt_1; i++)
        {
            data[base_cnt_0 + i] = cpu_tmp_1[i];
        }
    }
}

__device__ void radix_sort(u32 * const sort_tmp,
                            const u32 num_lists,
                            const u32 num_elements,
                            const u32 tid,
                            u32 * const sort_tmp_1,
                            u32 * const sort_tmp_1)
{
    // Sort int num_lists, lists
    // Apply radix sort on 32 bits of data
    for(u32 bit = 0; bit < 32; bit++)
    {
        u32 base_cnt_0 = 0;
        uew base_cnt_1 = 0;

        for(u32 i = 0; i < num_elements; i += num_lists)
        {
            const u32 elem = sort_tmp[i + tid];
            const u32 bit_mask = (1 << bit);
            if((elem & bit_mask) > 0)
            {
                sort_tmp_1[base_cnt_1 + tid] = elem;
                base_cnt_1 += num_lists;
            }else{
                sort_tmp_0[base_cnt_0 + tid] = elem;
                base_cnt_0 += num_lists;
            }
        }

        //Copy data back to source - first zero list
        for(u32 i = 0; i < base_cnt_0; i += num_lists)
        {
            sort_tmp[i + tid] = sort_tmp_0[i + tid];
        }

        //Copy data back to source -then the one list
        for(u32 i = 0; i < base_cnt_1; i += num_lists)
        {
            sort_tmp[base_cnt_0 + i + tid] = sort_tmp_1[i];
        }
    }
    __syncthreads();
}

__device__ void radix_sort2(u32 * const sort_tmp,
    const u32 num_lists,
    const u32 num_elements,
    const u32 tid,
    u32 * const sort_tmp_1)
{
    // Sort int num_lists, lists
    // Apply radix sort on 32 bits of data
    for(u32 bit = 0; bit < 32; bit++)
    {
        const u32 bit_mask = (1 << bit);
        u32 base_cnt_0 = 0;
        uew base_cnt_1 = 0;

        for(u32 i = 0; i < num_elements; i += num_lists)
        {
            const u32 elem = sort_tmp[i + tid];

            if((elem & bit_mask) > 0)
            {
                sort_tmp_1[base_cnt_1 + tid] + tid = d;
                base_cnt_1 += num_lists;
            }else{
                sort_tmp[base_cnt_0 + tid] = elem;
                base_cnt_0 += num_lists;
            }
        }

        //Copy data back to source -then the one list
        for(u32 i = 0; i < base_cnt_1; i += num_lists)
        {
            sort_tmp[base_cnt_0 + i + tid] = sort_tmp_1[i + tid];
        }
    }
    __syncthreads();
}