
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(int *a, int *b, int *c, int n)
{
    //int i = threadIdx.x;
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    c[i] = a[i] + b[i];
}

int main(void)
{
    int *a, *b, *c;
    int SIZE = 1024;
    //alloc the memory
    hipMallocManaged(&a, SIZE * sizeof(int));
    hipMallocManaged(&b, SIZE * sizeof(int));
    hipMallocManaged(&c, SIZE * sizeof(int));
    //fill the array
    for(int i = 0; i < SIZE; i++)
    {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }
    //call the kernel function
    vectorAdd<<<2, SIZE/2>>>(a, b, c, SIZE);
    hipDeviceSynchronize();

    for(int i = 0; i < SIZE; i++)
    {
        printf("c[%d] = %d\n", i, c[i]);
    }
    //release function
    hipFree(a);
    hipFree(b);
    hipFree(c);

    return 0;
}
